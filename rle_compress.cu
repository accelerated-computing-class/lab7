#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <random>
#include <vector>

////////////////////////////////////////////////////////////////////////////////
// Utility Functions

void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

////////////////////////////////////////////////////////////////////////////////
// CPU Reference Implementation (Already Written)

void rle_compress_cpu(
    uint32_t raw_count,
    char const *raw,
    std::vector<char> &compressed_data,
    std::vector<uint32_t> &compressed_lengths) {
    compressed_data.clear();
    compressed_lengths.clear();

    uint32_t i = 0;
    while (i < raw_count) {
        char c = raw[i];
        uint32_t run_length = 1;
        i++;
        while (i < raw_count && raw[i] == c) {
            run_length++;
            i++;
        }
        compressed_data.push_back(c);
        compressed_lengths.push_back(run_length);
    }
}

/// <--- your code here --->

////////////////////////////////////////////////////////////////////////////////
// Optimized GPU Implementation

namespace rle_gpu {

/* TODO: your GPU kernels here... */

// Returns desired size of scratch buffer in bytes.
size_t get_workspace_size(uint32_t raw_count) {
    /* TODO: your CPU code here... */
    return 0;
}

// 'launch_rle_compress'
//
// Input:
//
//   'raw_count': Number of bytes in the input buffer 'raw'.
//
//   'raw': Uncompressed bytes in GPU memory.
//
//   'workspace': Scratch buffer in GPU memory. The size of the scratch buffer
//   in bytes is determined by 'get_workspace_size'.
//
// Output:
//
//   Returns: 'compressed_count', the number of runs in the compressed data.
//
//   'compressed_data': Output buffer of size 'raw_count' in GPU memory. The
//   function should fill the first 'compressed_count' bytes of this buffer
//   with the compressed data.
//
//   'compressed_lengths': Output buffer of size 'raw_count' in GPU memory. The
//   function should fill the first 'compressed_count' integers in this buffer
//   with the lengths of the runs in the compressed data.
//
uint32_t launch_rle_compress(
    uint32_t raw_count,
    char const *raw,             // pointer to GPU buffer
    void *workspace,             // pointer to GPU buffer
    char *compressed_data,       // pointer to GPU buffer
    uint32_t *compressed_lengths // pointer to GPU buffer
) {
    /* TODO: your CPU code here... */
    uint32_t compressed_count = 0;
    return compressed_count;
}

} // namespace rle_gpu

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

template <typename Reset, typename F>
double benchmark_ms(double target_time_ms, Reset &&reset, F &&f) {
    double best_time_ms = std::numeric_limits<double>::infinity();
    double elapsed_ms = 0.0;
    while (elapsed_ms < target_time_ms) {
        reset();
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        f();
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double this_ms = std::chrono::duration<double, std::milli>(end - start).count();
        elapsed_ms += this_ms;
        best_time_ms = std::min(best_time_ms, this_ms);
    }
    return best_time_ms;
}

struct Results {
    double time_ms;
};

enum class Mode {
    TEST,
    BENCHMARK,
};

Results run_config(Mode mode, std::vector<char> const &raw) {
    // Allocate buffers
    size_t workspace_size = rle_gpu::get_workspace_size(raw.size());
    char *raw_gpu;
    void *workspace;
    char *compressed_data_gpu;
    uint32_t *compressed_lengths_gpu;
    CUDA_CHECK(hipMalloc(&raw_gpu, raw.size()));
    CUDA_CHECK(hipMalloc(&workspace, workspace_size));
    CUDA_CHECK(hipMalloc(&compressed_data_gpu, raw.size()));
    CUDA_CHECK(hipMalloc(&compressed_lengths_gpu, raw.size() * sizeof(uint32_t)));

    // Copy input data to GPU
    CUDA_CHECK(hipMemcpy(raw_gpu, raw.data(), raw.size(), hipMemcpyHostToDevice));

    auto reset = [&]() {
        CUDA_CHECK(hipMemset(compressed_data_gpu, 0, raw.size()));
        CUDA_CHECK(hipMemset(compressed_lengths_gpu, 0, raw.size() * sizeof(uint32_t)));
    };

    auto f = [&]() {
        rle_gpu::launch_rle_compress(
            raw.size(),
            raw_gpu,
            workspace,
            compressed_data_gpu,
            compressed_lengths_gpu);
    };

    // Test correctness
    reset();
    uint32_t compressed_count = rle_gpu::launch_rle_compress(
        raw.size(),
        raw_gpu,
        workspace,
        compressed_data_gpu,
        compressed_lengths_gpu);
    std::vector<char> compressed_data(compressed_count);
    std::vector<uint32_t> compressed_lengths(compressed_count);
    CUDA_CHECK(hipMemcpy(
        compressed_data.data(),
        compressed_data_gpu,
        compressed_count,
        hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(
        compressed_lengths.data(),
        compressed_lengths_gpu,
        compressed_count * sizeof(uint32_t),
        hipMemcpyDeviceToHost));

    std::vector<char> compressed_data_expected;
    std::vector<uint32_t> compressed_lengths_expected;
    rle_compress_cpu(
        raw.size(),
        raw.data(),
        compressed_data_expected,
        compressed_lengths_expected);

    bool correct = true;
    if (compressed_count != compressed_data_expected.size()) {
        printf("Mismatch in compressed count:\n");
        printf("  Expected: %zu\n", compressed_data_expected.size());
        printf("  Actual:   %u\n", compressed_count);
        correct = false;
    }
    if (correct) {
        for (size_t i = 0; i < compressed_data_expected.size(); i++) {
            if (compressed_data[i] != compressed_data_expected[i]) {
                printf("Mismatch in compressed data at index %zu:\n", i);
                printf(
                    "  Expected: 0x%02x\n",
                    static_cast<unsigned char>(compressed_data_expected[i]));
                printf(
                    "  Actual:   0x%02x\n",
                    static_cast<unsigned char>(compressed_data[i]));
                correct = false;
                break;
            }
            if (compressed_lengths[i] != compressed_lengths_expected[i]) {
                printf("Mismatch in compressed lengths at index %zu:\n", i);
                printf("  Expected: %u\n", compressed_lengths_expected[i]);
                printf("  Actual:   %u\n", compressed_lengths[i]);
                correct = false;
                break;
            }
        }
    }
    if (!correct) {
        if (raw.size() <= 1024) {
            printf("\nInput:\n");
            for (size_t i = 0; i < raw.size(); i++) {
                printf("  [%4zu] = 0x%02x\n", i, static_cast<unsigned char>(raw[i]));
            }
            printf("\nExpected:\n");
            for (size_t i = 0; i < compressed_data_expected.size(); i++) {
                printf(
                    "  [%4zu] = data: 0x%02x, length: %u\n",
                    i,
                    static_cast<unsigned char>(compressed_data_expected[i]),
                    compressed_lengths_expected[i]);
            }
            printf("\nActual:\n");
            if (compressed_data.size() == 0) {
                printf("  (empty)\n");
            }
            for (size_t i = 0; i < compressed_data.size(); i++) {
                printf(
                    "  [%4zu] = data: 0x%02x, length: %u\n",
                    i,
                    static_cast<unsigned char>(compressed_data[i]),
                    compressed_lengths[i]);
            }
        }
        exit(1);
    }

    if (mode == Mode::TEST) {
        return {};
    }

    // Benchmark
    double target_time_ms = 1000.0;
    double time_ms = benchmark_ms(target_time_ms, reset, f);

    // Cleanup
    CUDA_CHECK(hipFree(raw_gpu));
    CUDA_CHECK(hipFree(workspace));
    CUDA_CHECK(hipFree(compressed_data_gpu));
    CUDA_CHECK(hipFree(compressed_lengths_gpu));

    return {time_ms};
}

template <typename Rng> std::vector<char> generate_test_data(uint32_t size, Rng &rng) {
    auto random_byte = std::uniform_int_distribution<int32_t>(
        std::numeric_limits<char>::min(),
        std::numeric_limits<char>::max());
    constexpr uint32_t alphabet_size = 4;
    auto alphabet = std::vector<char>();
    for (uint32_t i = 0; i < alphabet_size; i++) {
        alphabet.push_back(random_byte(rng));
    }
    auto random_symbol = std::uniform_int_distribution<uint32_t>(0, alphabet_size - 1);
    auto data = std::vector<char>();
    for (uint32_t i = 0; i < size; i++) {
        data.push_back(alphabet.at(random_symbol(rng)));
    }
    return data;
}

int main(int argc, char const *const *argv) {
    auto rng = std::mt19937(0xCA7CAFE);

    auto test_sizes = std::vector<uint32_t>{
        16,
        10,
        128,
        100,
        1 << 10,
        1000,
        1 << 20,
        1'000'000,
        16 << 20,
    };

    printf("Correctness:\n\n");
    for (auto test_size : test_sizes) {
        auto raw = generate_test_data(test_size, rng);
        printf("  Testing compression for size %u\n", test_size);
        run_config(Mode::TEST, raw);
        printf("  OK\n\n");
    }

    auto test_data_search_paths = std::vector<std::string>{".", "/"};
    std::string test_data_path;
    for (auto test_data_search_path : test_data_search_paths) {
        auto candidate_path = test_data_search_path + "/rle_raw.bmp";
        if (std::filesystem::exists(candidate_path)) {
            test_data_path = candidate_path;
            break;
        }
    }
    if (test_data_path.empty()) {
        printf("Could not find test data file.\n");
        exit(1);
    }

    auto raw = std::vector<char>();
    {
        auto file = std::ifstream(test_data_path, std::ios::binary);
        if (!file) {
            printf("Could not open test data file '%s'.\n", test_data_path.c_str());
            exit(1);
        }
        file.seekg(0, std::ios::end);
        raw.resize(file.tellg());
        file.seekg(0, std::ios::beg);
        file.read(raw.data(), raw.size());
    }

    printf("Performance:\n\n");
    printf("  Testing compression on file 'rle_raw.bmp' (size %zu)\n", raw.size());
    auto results = run_config(Mode::BENCHMARK, raw);
    printf("  Time: %.2f ms\n", results.time_ms);

    return 0;
}